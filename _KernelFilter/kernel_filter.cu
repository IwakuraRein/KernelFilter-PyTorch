
#include <hip/hip_runtime.h>
__global__ void KernelFilterKernel(
	int grid_count,
	const float* grid, 
	const float* kernel,
	const int batch,
	const int height,
	const int width,
	const int channel,
	const int k0, // filter size
	const int half_k, // k0 / 2
	const int dilation,
	float* output
) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < grid_count; i += blockDim.x * gridDim.x)
	{
		const int w = i % width;
		const int h = (i / width) % height;
		const int c = (i / (width * height)) % channel;
		const int b = (i / (channel * width * height)) % batch;
		const int k_sq = k0 * k0;
		// const int kernel_center = (k_sq - 1) / 2

		// const int sx = 1;
		const int sy = width;
		const int sc = width * height;
		const int sb = channel * width * height;

		float out_value = 0.0f;
		float out_weight = 0.0f;

		
		const int kernel_base = w + h*sy + b * k_sq * width * height; // find filter's weight
		for (int ii_o = -half_k; ii_o <= half_k; ii_o++)
		{
			int xx_o = w + ii_o*(dilation);
			if (xx_o < 0 || xx_o > width - 1)
				continue;
			for (int jj_o = -half_k; jj_o <= half_k; jj_o++)
			{
				int yy_o = h + jj_o*(dilation);
				if (yy_o < 0 || yy_o > height - 1)
					continue;
				int kernel_idx = ((ii_o + half_k) + (jj_o + half_k) * k0)*sc + kernel_base;
				int grid_idx = c*sc + xx_o + yy_o * sy + b * sb;
				if (grid[grid_idx] > 0.0f)
				{
					out_value += grid[grid_idx] * kernel[kernel_idx];
					out_weight += kernel[kernel_idx];
				}
			}	
		}
		output[i] = out_value / (out_weight + 1e-8f);
		// output[i] = h*1000+w;
	}
}

__global__ void KernelFilterGridGradKernel(
	int grid_count,
	const float* grid, 
	const float* kernel,
	const float* backprop,
	const int batch,
	const int height,
	const int width,
	const int channel,
	const int k0,
	const int half_k,
	const int dilation,
	float* grid_grad
) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < grid_count; i += blockDim.x * gridDim.x)
	{	
		// compute the location (b, h, w, c) of this pixel
		const int w = i % width;
		const int h = (i / width) % height;
		const int c = (i / (width * height)) % channel;
		const int b = (i / (channel * width * height)) % batch;
		const int k_sq = k0 * k0;
		// const int kernel_center = (k_sq - 1) / 2

		// scales
		// const int sx = 1;
		const int sy = width;
		const int sc = width*height;
		const int sb = channel * width * height;

		float out_value = 0.0f;
		const int kernel_base = k_sq * width * height * b; // batch offset
		
		for (int ii_o = -half_k; ii_o <= half_k; ii_o++)
		{
			int xx_o = w + ii_o*(dilation);
			if (xx_o < 0 || xx_o > width - 1)
				continue;
			for (int jj_o = -half_k; jj_o <= half_k; jj_o++)
			{
				int yy_o = h + jj_o*(dilation); // (xx_o, yy_o) is a filter's center position
				if (yy_o < 0 || yy_o > height - 1)
					continue;
				// the value of weight (-ii_o, -jj_o) of the filter (xx_o, yy_o)
				const int kernel_base_2 = xx_o + sy * yy_o + kernel_base;
				int kernel_idx = ((-ii_o + half_k) + (-jj_o + half_k) * k0)*sc + kernel_base_2;        
				
				float part1 = grid[i] > 0 ? kernel[kernel_idx] : 0;
				float part2 = 0;

				for (int ii_i = -half_k; ii_i <= half_k; ii_i++) // compute ∑weight of this filter
				{
					// (xx_i, yy_i) is the location of pixel match the weight (ii_i, jj_i)
					int xx_i = xx_o + ii_i*(dilation);
					if (xx_i < 0 || xx_i > width - 1)
						continue;
					for(int jj_i = -half_k; jj_i <= half_k; jj_i++)
					{
						int yy_i = yy_o + jj_i*(dilation);
						if (yy_i < 0 || yy_i > height - 1)
							continue;
						int grid_idx_i = xx_i + yy_i * sy + c * sc + b * sb;
						int kernel_idx_i = (ii_i + half_k + (jj_i + half_k) * k0)*sc + kernel_base_2;        
						if (grid[grid_idx_i] > 0)
						{
							part2 += kernel[kernel_idx_i];
						}
					}
				}
				int grid_idx_bp = xx_o + yy_o * sy + c * sc + b * sb;
				out_value += backprop[grid_idx_bp] * part1 / (part2 + 1e-8f);
			}
		}
		grid_grad[i] = out_value;
	}
}

__global__ void KernelFilterKernelGradKernel(
	int weight_count,
	const float* grid, 
	const float* kernel,
	const float* backprop,
	const int batch,
	const int height,
	const int width,
	const int channel,
	const int k0,
	const int half_k,
	const int dilation,
	float* kernel_grad
) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < weight_count; i += blockDim.x * gridDim.x)
	{	
		int k_sq = k0 * k0;

		// compute the center location (b, h, w, k) of this filter
		const int w = i % width;
		const int h = (i / width) % height;
		const int k = (i / (width * height)) % k_sq;
		const int b = (i / (width * height * k_sq)) % batch;
		// const int kernel_center = (k_sq - 1) / 2

		// scale
		//const int sx = 1;
		const int sy = width;
		const int sc = width*height;
		const int sb = channel * width * height;
		
		float out_value = 0.0f;
		const int kernel_base = i - k*sc; // the index of the first weight of this filter
		
		// (xx_o, yy_o) is the grid pixel that multiplied with this kernel_weight
		int xx_o = w + (k % k0 - half_k)*(dilation);
		int yy_o = h + (k / k0 - half_k)*(dilation);
		if (xx_o < 0 || xx_o > width - 1 || yy_o < 0 || yy_o > height - 1)
			out_value = 0.0f;
		else
		{
			for (int c = 0; c < channel; c++)
			{
				int grid_idx = xx_o + sy * yy_o + sc*c + sb * b; // the pixel that multiplied with this kernel_weight
				float part1 = grid[grid_idx];
				float part2 = 0.0f; // store the ∑weight of this filter
				float part3 = 0.0f; // store the filtered pixel
				if (part1 > 0)
				{
					for (int ii_i = -half_k; ii_i <= half_k; ii_i++)
					{
						int xx_i = w + ii_i*(dilation);
						if (xx_i < 0 || xx_i > width - 1)
							continue;
						for (int jj_i = -half_k; jj_i <= half_k; jj_i++)
						{
							int yy_i = h + jj_i*(dilation);
							if (yy_i < 0 || yy_i > height - 1)
								continue;
							int grid_idx_i = xx_i + yy_i * sy + c * sc + b * sb;
							int kernel_idx_i = (ii_i + half_k + (jj_i + half_k) * k0)*sc + kernel_base; 
							if (grid[grid_idx_i] > 0)
							{
								part3 += grid[grid_idx_i] * kernel[kernel_idx_i];
								part2 += kernel[kernel_idx_i];
							}
						}
					}
				int grid_idx_bp = w + sy * h + sc * c + sb * b;
				out_value += (part1 * part2 - part3) / (part2 * part2 + 1e-8f) * backprop[grid_idx_bp];
				}
			}
		}
		kernel_grad[i] = out_value;
	}
}

void KernelFilterKernelLauncher(
	const float* grid,
	const float* kernel,
	const int dilation,
	const int* grid_size,
	const int* kernel_size,
	float* output
) {
	int batch = grid_size[0];
	int channel = grid_size[1];
	int height = grid_size[2];
	int width = grid_size[3];

	int k0 = sqrt(kernel_size[1]);
	int half_k = k0 / 2;

	int grid_count = batch * height * width * channel;
	if (grid_count > 0) {
		dim3 GRID((grid_count + 1023) / 1024);
		dim3 BLOCK(1024);
		KernelFilterKernel<<<GRID, BLOCK>>>(
			grid_count,
			grid,
			kernel,
			batch,
			height,
			width,
			channel,
			k0,
			half_k,
			dilation,
			output);
	}
}

void KernelFilterGradKernelLauncher(
	const float* grid,
	const float* kernel,
	const int dilation,
	const float* backprop,
	const int* grid_size,
	const int* kernel_size,
	float* grid_grad,
	float* kernel_grad
) {
	int batch = grid_size[0];
	int channel = grid_size[1];
	int height = grid_size[2];
	int width = grid_size[3];

	int k0 = sqrt(kernel_size[1]);
	int half_k = k0 / 2;


	int grid_count = batch * height * width * channel;
	if (grid_count > 0) {
		dim3 GRID((grid_count + 1023) / 1024);
		dim3 BLOCK(1024);
		KernelFilterGridGradKernel<<<GRID, BLOCK>>>(
			grid_count,
			grid,
			kernel,
			backprop,
			batch,
			height,
			width,
			channel,
			k0,
			half_k,
			dilation,
			grid_grad);
	}

	int weight_count = batch * height * width * kernel_size[1];
	if(weight_count > 0) {
		dim3 GRID((grid_count + 1023) / 1024);
		dim3 BLOCK(1024);
		KernelFilterKernelGradKernel<<<GRID, BLOCK>>>(
			weight_count,
			grid,
			kernel,
			backprop,
			batch,
			height,
			width,
			channel,
			k0,
			half_k,
			dilation,
			kernel_grad);
	}
}